#include "hip/hip_runtime.h"
#include "headers/convolution.h"

// set kernel as average kernel
void fill_mean_kernel (matrix K, const int K_DIM){
    matrix_element mean = 1.0 / (K_DIM * K_DIM);

    for(int i = 0; i < K_DIM; i++){
        for(int j = 0; j < K_DIM; j++){
            K[i*K_DIM + j] = mean;
        }
    }
}
// Get the number of bytes read and written by the convolution kernel
long int get_conv_bytes_read_write(const int W, const int H, const int C, const int PAD, const int K_DIM){
    long int TOT_SIZE_NOPAD = W * H * C; 
    long int TOT_K_DIM = K_DIM*K_DIM;
    
    long int BR = sizeof(matrix_element) * ((TOT_K_DIM + TOT_K_DIM) * TOT_SIZE_NOPAD); 
    long int BW = sizeof(matrix_element) * TOT_SIZE_NOPAD;

    return BR + BW;
}
// Get the number of FLOPs performed by the convolution kernel
long int get_conv_flops(const int W, const int H, const int C, const int PAD, const int K_DIM){
    long int TOT_SIZE_NOPAD = W * H * C; 
    long int TOT_K_DIM = K_DIM*K_DIM;
    
    long int FLOP = (TOT_K_DIM + TOT_K_DIM) * TOT_SIZE_NOPAD;

    return FLOP;
}


// ================================================================================================================================= //
// ================================================== CONVOLUTION IMPLEMENTATIONS ================================================== //
// ================================================================================================================================= //


void cpu_convolution_naive(matrix image, matrix K, matrix output, const int W, const int H, const int C, const int PAD, const int K_DIM){
    int u,v;               // image pixel indeces (on which conv. is currently computed)
    int i,j;               // kernel indeces
    int patch_u, patch_v;  // image pixel currently evaluated by kernel
    int c;                 // channel index
    const int K_CENTER = (int)(K_DIM / 2);
    matrix_element sum;

    for(u = PAD; u < (H + PAD); u++){
        for(v = PAD; v < (W + PAD); v++){
	        for(c = 0; c < C; c++){
                sum = 0.0;
                for(i = 0; i < K_DIM; i++){
                    for(j = 0; j < K_DIM; j++){
                        patch_u = u - K_CENTER + i;
                        patch_v = v - K_CENTER + j;
                    
                        sum += K[i*K_DIM + j] * image[patch_u*(W + 2*PAD)*C + patch_v*C + c];
                    }   
                }
                output[(u-PAD)*W*C + (v-PAD)*C + c] = sum;
            }
        }
     }
}


__global__
void gpu_convolution_naive(matrix image, matrix K, matrix output, const int W, const int H, const int C, const int PAD, const int K_DIM){
    int u = blockIdx.x*blockDim.x + threadIdx.x; // image pixel (u) (on which conv. is currently computed)
    int v = blockIdx.y*blockDim.y + threadIdx.y; // image pixel (v) (on which conv. is currently computed)
    
    if(u >= PAD && v >= PAD && u < W+PAD && v < H+PAD){
        int i,j;                // kernel indeces
        int patch_u, patch_v;   // image pixel currently evaluated by kernel
        int c;                  // channel index
        const int K_CENTER = (int)(K_DIM / 2);
        matrix_element sum;

        for(c = 0; c < C; c++){
            sum = 0.0;
            for(i = 0; i < K_DIM; i++){
                for(j = 0; j < K_DIM; j++){
                        patch_u = u - K_CENTER + i;
                        patch_v = v - K_CENTER + j;

                        sum += K[i*K_DIM + j] * image[patch_u*(W + 2*PAD)*C + patch_v*C + c];
                    }
                }
            output[(u-PAD)*W*C + (v-PAD)*C + c] = sum;
        }
    }
}

__global__ 
void gpu_convolution_shared(matrix image, matrix K, matrix output, const int W, const int H, const int C, const int PAD, const int K_DIM) {
    int u = blockIdx.x*blockDim.x + threadIdx.x; // image pixel (u) (on which conv. is currently computed)
    int v = blockIdx.y*blockDim.y + threadIdx.y; // image pixel (v) (on which conv. is currently computed)

    extern __shared__ matrix_element shared_image[];

    if(u < W+2*PAD && v < H+2*PAD){
        for(int c = 0; c < C; c++){
            shared_image[((threadIdx.x+PAD)*blockDim.x)*C + (threadIdx.y+PAD)*C + c] = image[u*(W + 2*PAD)*C + v*C + c]; //Maybe remove padding
        }
    }
    
    if(u >= PAD && v >= PAD && u < W+PAD && v < H+PAD){
        // Thread of left edge must also copy left padding
        if(threadIdx.x < PAD){
            for(int c = 0; c < C; c++){
                shared_image[threadIdx.x*blockDim.x*C + (threadIdx.y+PAD)*C + c] = image[(u - PAD)*(W + 2*PAD)*C + v*C + c];
            }
        }

        // Thread of top edge must also copy top padding
        if(threadIdx.y < PAD){
            for(int c = 0; c < C; c++){
                shared_image[(threadIdx.x+PAD)*blockDim.x*C + threadIdx.y*C + c] = image[u*(W + 2*PAD)*C +(v-PAD)*C + c];
            }
        }
        // Thread of right edge must also copy right padding
        if(threadIdx.x + PAD > blockDim.x){
            for(int c = 0; c < C; c++){
                shared_image[(threadIdx.x + 2*PAD)*blockDim.x*C + (threadIdx.y+PAD)*C + c] = image[(u + PAD)*(W + 2*PAD)*C + v*C + c];
            }
        }
        // Thread of bottom edge must also copy bottom padding
        if(threadIdx.y + PAD > blockDim.y){
            for(int c = 0; c < C; c++){
                shared_image[(threadIdx.x+PAD)*blockDim.x*C + (threadIdx.y + 2*PAD)*C + c] = image[u*(W + 2*PAD)*C +(v+PAD)*C + c];
            }
        }
    }
    
    // Thread for corner 
    // Thread for corner 
    // Thread for corner 
    // Thread for corner 

    __syncthreads();

    if(u >= PAD && v >= PAD && u < W+PAD && v < H+PAD){
        int i,j;                // kernel indeces
        int patch_u, patch_v;   // image pixel currently evaluated by kernel
        int c;                  // channel index
        const int K_CENTER = (int)(K_DIM / 2);
        matrix_element sum;

        for(c = 0; c < C; c++){
            sum = 0.0;
            for(i = 0; i < K_DIM; i++){
                for(j = 0; j < K_DIM; j++){
                        patch_u = threadIdx.x + 2*PAD - K_CENTER + i;
                        patch_v = threadIdx.y + 2*PAD - K_CENTER + j;

                        sum += K[i*K_DIM + j] * shared_image[patch_u*C*blockDim.x + patch_v*C + c];
                    }
                }
            output[(u-PAD)*W*C + (v-PAD)*C + c] = sum;
        }
    }
}
