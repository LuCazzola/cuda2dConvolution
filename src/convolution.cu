#include "hip/hip_runtime.h"
#include "headers/convolution.h"

__global__
void gpu_convolution(int image_dim_x, int image_dim_y, int* image, int K_dim, float* K, int* output){
    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    float sum = 0.0;
    int K_center = K_dim / 2;

    if(thread_x >= 0 && thread_x < image_dim_x && thread_y >= 0 && thread_y < image_dim_y){
        sum = 0.0;
        for(int i = 0; i < K_dim; i++){
            for(int j = 0; j < K_dim; j++){
                int patch_i = thread_x-K_center+i;
                int patch_j = thread_y-K_center+j;
                if(patch_i < 0 || patch_i >= image_dim_x || patch_j < 0 || patch_j >= image_dim_y){
                    sum += 0.0;
                }
                else {
                    sum += K[i*K_dim+j] * (float) image[patch_i*image_dim_x+patch_j];
                }
            }
        }
        output[thread_x*image_dim_x+thread_y] = (int) sum;
    }
}

void cpu_convolution(PngImage* image, int K_dim, matrix K, PngImage* output){
    unsigned int u,v;               // image pixel indeces (on which conv. is currently computed)
    unsigned int i,j;               // kernel indeces
    unsigned int patch_u, patch_v;  // image pixel currently evaluated by kernel
    unsigned int c;                 // channel index
    
    matrix_element sum = 0.0;
    unsigned int K_center = K_dim / 2;

    for(u = image->PAD; u < (image->H + image->PAD); u++){
        for(v = image->PAD; v < (image->W + image->PAD); v++){
	    for(c = 0; c < image->C; c++){
                sum = 0.0;
                for(i = 0; i < K_dim; i++){
                    for(j = 0; j < K_dim; j++){
                        patch_u = u-K_center+i;
                        patch_v = v-K_center+j;
                    
                        sum += K[i*K_dim+j] * image->val[patch_u * (image->W + 2*image->PAD) * image->C +patch_v*image->C + c];
                    }   
                }
                output->val[u*image->W + v + c] = sum;
            }
        }
     }
}
