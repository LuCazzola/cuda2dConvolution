#include "hip/hip_runtime.h"
#include "headers/convolution.h"

// set kernel as average kernel
void fill_mean_kernel (matrix K, const int K_DIM){
    matrix_element mean = 1.0 / (K_DIM * K_DIM);

    for(int i = 0; i < K_DIM; i++){
        for(int j = 0; j < K_DIM; j++){
            K[i*K_DIM + j] = mean;
        }
    }
}
// Get the number of bytes read and written by the convolution kernel
long int get_conv_bytes_read_write(const int W, const int H, const int C, const int K_DIM){
    long int TOT_SIZE = W * H * C; 
    long int TOT_K_DIM = K_DIM*K_DIM;
    
    long int BR = sizeof(matrix_element) * ((TOT_K_DIM + TOT_K_DIM) * TOT_SIZE); 
    long int BW = sizeof(matrix_element) * TOT_SIZE;

    return BR + BW;
}
// Get the number of FLOPs performed by the convolution kernel
long int get_conv_flops(const int W, const int H, const int C, const int K_DIM){
    long int TOT_SIZE = W * H * C; 
    long int TOT_K_DIM = K_DIM*K_DIM;
    
    long int FLOP = (TOT_K_DIM + TOT_K_DIM) * TOT_SIZE;

    return FLOP;
}


// ================================================================================================================================= //
// ================================================== CONVOLUTION IMPLEMENTATIONS ================================================== //
// ================================================================================================================================= //


void cpu_convolution_naive(matrix image, matrix K, matrix output, const int W, const int H, const int C, const int K_DIM){
    int u,v;               // image pixel indeces (on which conv. is currently computed)
    int i,j;               // kernel indeces
    int patch_u, patch_v;  // image pixel currently evaluated by kernel
    int c;                 // channel index
    const int K_CENTER = (int)(K_DIM / 2);
    matrix_element sum;

    for(u = 0; u < H; u++){
        for(v = 0; v < W; v++){
	        for(c = 0; c < C; c++){
                sum = 0.0;
                for(i = 0; i < K_DIM; i++){
                    for(j = 0; j < K_DIM; j++){
                        patch_u = u - K_CENTER + i;
                        patch_v = v - K_CENTER + j;
                        if (patch_u >= 0 && patch_v >= 0 && patch_u < W && patch_v < H){
                            sum += K[i*K_DIM + j] * image[patch_u*W*C + patch_v*C + c];
                        }
                    }   
                }
                output[u*W*C + v*C + c] = sum;
            }
        }
     }
}


__global__
void gpu_convolution_naive(matrix image, matrix K, matrix output, const int W, const int H, const int C, const int K_DIM){
    int u = blockIdx.x*blockDim.x + threadIdx.x; // image pixel (u) (on which conv. is currently computed)
    int v = blockIdx.y*blockDim.y + threadIdx.y; // image pixel (v) (on which conv. is currently computed)
    
    if(u >= 0 && v >= 0 && u < W && v < H){
        int i,j;                // kernel indeces
        int patch_u, patch_v;   // image pixel currently evaluated by kernel
        int c;                  // channel index
        const int K_CENTER = (int)(K_DIM / 2);
        matrix_element sum;

        for(c = 0; c < C; c++){
            sum = 0.0;
            for(i = 0; i < K_DIM; i++){
                for(j = 0; j < K_DIM; j++){
                        patch_u = u - K_CENTER + i;
                        patch_v = v - K_CENTER + j;
                        if (patch_u >= 0 && patch_v >= 0 && patch_u < W && patch_v < H){
                            sum += K[i*K_DIM + j] * image[patch_u*W*C + patch_v*C + c];
                        }
                    }
                }
            output[u*W*C + v*C + c] = sum;
        }
    }
}

__global__ 
void gpu_convolution_shared(matrix image, matrix K, matrix output, const int W, const int H, const int C, const int K_DIM) {
    
    const int PAD = (int) (K_DIM / 2);
    const int IN_TILE_DIM = blockDim.x;
    const int OUT_TILE_DIM = IN_TILE_DIM - 2*PAD;
    
    int col = blockIdx.x*OUT_TILE_DIM + threadIdx.x - PAD;
    int row = blockIdx.y*OUT_TILE_DIM + threadIdx.y - PAD; 

    // Loading input tile
    extern __shared__ matrix_element buffer[];
    matrix shared_image = &buffer[0];
    matrix shared_K = &buffer[(IN_TILE_DIM + 2*PAD)*(IN_TILE_DIM + 2*PAD)*C];

    // Loading kernel
    if (threadIdx.x == 0 && threadIdx.y == 0){
        for(int i = 0; i < K_DIM; i++){
            for(int j = 0; j < K_DIM; j++){
                shared_K[i*K_DIM + j] = K[i*K_DIM + j];
            }
        }
    }
    // Load image patch
    for(int c = 0; c < C; c++){
        if(row >= 0 && row < H && col >= 0 && col < W){
            shared_image[threadIdx.y*blockDim.x*C + threadIdx.x*C + c] = image[row*W*C + col*C + c];
        }
        else {
            shared_image[threadIdx.y*blockDim.x*C + threadIdx.x*C + c] = 0.0;
        }
    }

    __syncthreads();

    // Calculating output elements
    int tileCol = threadIdx.x - PAD;
    int tileRow = threadIdx.y - PAD;
    // Turning off threads at the edges of the block
    if(col >= 0 && col < W && row >= 0 && row < H){
        if(tileCol >= 0 && tileCol < OUT_TILE_DIM && tileRow >= 0 && tileRow < OUT_TILE_DIM){
            for(int c = 0; c < C; c++){
                float sum = 0.0f;
                for(int i = 0; i < K_DIM; i++){
                    for(int j = 0; j < K_DIM; j++){
                        sum += shared_K[i*K_DIM + j] * shared_image[(tileRow+i)*blockDim.x*C + (tileCol+j)*C + c];
                    }
                }
                output[row*W*C + col*C + c] = sum;
            }
        }
    }
}


// kernel in constant memory
__constant__ matrix_element c_k[CONST_MEM_SIZE];

void fill_const_kernel (matrix h_k, const int TOT_K_DIM){
    checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(c_k), h_k, TOT_K_DIM * sizeof(matrix_element), 0, hipMemcpyHostToDevice) );
}

__global__ 
void gpu_convolution_shared_constk(matrix image, matrix output, const int W, const int H, const int C, const int K_DIM) {
    
    const int PAD = (int) (K_DIM / 2);
    const int IN_TILE_DIM = blockDim.x;
    const int OUT_TILE_DIM = IN_TILE_DIM - 2*PAD;
    
    int col = blockIdx.x*OUT_TILE_DIM + threadIdx.x - PAD;
    int row = blockIdx.y*OUT_TILE_DIM + threadIdx.y - PAD; 

    // Loading input tile
    extern __shared__ matrix_element shared_image[];

    // Load image patch
    for(int c = 0; c < C; c++){
        if(row >= 0 && row < H && col >= 0 && col < W){
            shared_image[threadIdx.y*blockDim.x*C + threadIdx.x*C + c] = image[row*W*C + col*C + c];
        }
        else {
            shared_image[threadIdx.y*blockDim.x*C + threadIdx.x*C + c] = 0.0;
        }
    }

    __syncthreads();

    // Calculating output elements
    int tileCol = threadIdx.x - PAD;
    int tileRow = threadIdx.y - PAD;
    // Turning off threads at the edges of the block
    if(col >= 0 && col < W && row >= 0 && row < H){
        if(tileCol >= 0 && tileCol < OUT_TILE_DIM && tileRow >= 0 && tileRow < OUT_TILE_DIM){
            for(int c = 0; c < C; c++){
                float sum = 0.0f;
                for(int i = 0; i < K_DIM; i++){
                    for(int j = 0; j < K_DIM; j++){
                        sum += c_k[i*K_DIM + j] * shared_image[(tileRow+i)*blockDim.x*C + (tileCol+j)*C + c];
                    }
                }
                output[row*W*C + col*C + c] = sum;
            }
        }
    }
}

__global__ 
void gpu_convolution_shared_constk_cached(matrix image, matrix output, const int W, const int H, const int C, const int K_DIM) {
    const int PAD = (int) (K_DIM / 2);
    const int TILE_DIM = blockDim.x;
    
    int col = blockIdx.x*TILE_DIM + threadIdx.x;
    int row = blockIdx.y*TILE_DIM + threadIdx.y; 

    // Loading input tile
    extern __shared__ matrix_element shared_image[];

    // Load image patch
    for(int c = 0; c < C; c++){
        if(row < H && col < W){
            shared_image[threadIdx.y*TILE_DIM*C + threadIdx.x*C + c] = image[row*W*C + col*C + c];
        }
        else {
            shared_image[threadIdx.y*TILE_DIM*C + threadIdx.x*C + c] = 0.0;
        }
    }

    __syncthreads();


    // Calculating output elements
    int x = (int) threadIdx.x;
    int y = (int) threadIdx.y;
    if(col < W && row < H){
        for(int c = 0; c < C; c++){
            float sum = 0.0f;
            for(int i = 0; i < K_DIM; i++){
                for(int j = 0; j < K_DIM; j++){
                    if(x-PAD+j >= 0 && x-PAD+j < TILE_DIM && y-PAD+i >= 0 && y-PAD+i < TILE_DIM){
                        sum += c_k[i*K_DIM + j] * shared_image[(y+i-PAD)*TILE_DIM*C + (x+j-PAD)*C + c]; 
                    }
                    else if(row-PAD+i >= 0 && row-PAD+i < H && col-PAD+j >= 0 && col-PAD+j < W){
                        sum += c_k[i*K_DIM + j] * image[(row-PAD+i)*W*C + (col-PAD+j)*C + c];
                    }
                }
            }
            output[row*W*C + col*C + c] = sum;
        }
    }
}